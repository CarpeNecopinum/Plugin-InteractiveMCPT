#include "hip/hip_runtime.h"
#include "../MonteCuda.hh"
#include "../InfoStructs.hh"

#include <hip/hip_runtime.h>
#include <iostream>


/** CUDA Part **/

__global__ void fillWithOne(int* data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = 1;
}

__global__ void tracePixels(Point* pixels, float* output, mcMaterial* mats, mcTriangle* geometry, mcCameraInfo* cam)
{

}


/** C(++) Part **/

mcTriangle* devTriangles = 0;
mcMaterial* devMaterials = 0;
mcCameraInfo*   devCamera = 0;

void cudaTest(void)
{
    const size_t length = 64;
    int data[length];

    int* devPtr;
    hipMalloc(&devPtr, length * sizeof(int));

    fillWithOne<<<4, 16>>>(devPtr);

    hipMemcpy(data, devPtr, sizeof(int) * length, hipMemcpyDeviceToHost);

    for (size_t i = 0u; i < length; i++)
        std::cout << data[i] << std::endl;
}

void uploadBuffers(mcMaterial *materials, size_t materialCount, mcTriangle *tris, size_t triCount)
{
    hipFree(devMaterials);
    hipMalloc(&devMaterials, sizeof(mcMaterial) * materialCount);
    hipMemcpy(materials, devMaterials, sizeof(mcMaterial) * materialCount, hipMemcpyHostToDevice);

    hipFree(devTriangles);
    hipMalloc(&devTriangles, sizeof(mcTriangle) * triCount);
    hipMemcpy(tris, devTriangles, sizeof(mcTriangle) * triCount, hipMemcpyHostToDevice);
}

void uploadCameraInfo(const CameraInfo& cam)
{

    mcCameraInfo dev;
    for (int i = 0; i < 3; i++)
    {
        dev.eye_point[i] = (float) cam.eye_point[i];
        dev.image_plane_start[i] = (float) cam.image_plane_start[i];
        dev.x_dir[i] = (float) cam.x_dir[i];
        dev.y_dir[i] = (float) cam.y_dir[i];
    }

    hipFree(devCamera);
    hipMalloc(&devCamera, sizeof(mcCameraInfo));
    hipMemcpy(&dev, devCamera, sizeof(mcCameraInfo), hipMemcpyHostToDevice);
}


void cudaTracePixels(std::vector<Point> &pixels, RenderSettings settings, ACG::Vec3d* colorMap, uint32_t* sampleCounter, size_t imageWidth)
{
    Point* devPixels;
    hipMalloc(&devPixels, sizeof(Point) * pixels.size());
    hipMemcpy(pixels.data(), devPixels, sizeof(Point) * pixels.size(), hipMemcpyHostToDevice);

    float* devResults;
    hipMalloc(&devResults, sizeof(float) * 3 * pixels.size());

    tracePixels<<<pixels.size(),1>>>(devPixels, devResults, devMaterials, devTriangles, devCamera);
}

